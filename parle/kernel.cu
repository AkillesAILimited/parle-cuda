#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "helper_timer.h"
#include "hip/hip_runtime_api.h"

int parle(int *in, int n,
	int* h_symbolsOut,
	int* h_countsOut, int blockSize);


__global__ void scanKernel2(int *g_idata, int *g_odata, int n) {
	extern __shared__ int temp[]; // size is 2*n
	int tid = threadIdx.x;
	int pout = 0;
	int pin = 1;
	

	temp[pout*n + tid] = (tid > 0) ? g_idata[tid - 1] : 0;
	//temp[pout*n + i] = g_idata[i];

	__syncthreads();

	for (int offset = 1; offset < blockDim.x; offset *= 2)
	{
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (tid >= offset)
			temp[pout*n + tid] = temp[pin*n + tid] + temp[pin*n + tid - offset];
		else
			temp[pout*n + tid] = temp[pin*n + tid];
		__syncthreads();
	}

	g_odata[tid] = temp[pout*n + tid]; // write output
}


// segmented scan that is run on each thread block. 
__global__ void scanKernel(int *g_idata, int *g_odata, int n) {
	extern __shared__ int temp[]; // size is 2*n
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;
	int pout = 0;
	int pin = 1;



	temp[pout*n + i] = (tid > 0) ? g_idata[i-1] : 0;
	//temp[pout*n + i] = g_idata[i];

	__syncthreads();
	
	for (int offset = 1; offset < blockDim.x; offset *= 2)
	{
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (tid >= offset)
			temp[pout*n + i] = temp[pin*n + i] +  temp[pin*n + i - offset];
		else
			temp[pout*n + i] = temp[pin*n + i];
		__syncthreads();
	}
	
	g_odata[i] = temp[pout*n + i]; // write output
}

__global__ void getBlocksRunCountKernel(int *g_idata, int *g_odata, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){

		if (tid == (blockDim.x - 1)){
			g_odata[blockIdx.x] = g_idata[i]+1;
		}

	}
}

__global__ void scatterKernel(
	int *g_backwardMask, int* g_scannedBackwardMask, 
	int *g_forwardMask, int* g_scannedForwardMask,
	int *g_blocksOffset, int *g_in, 
	int *g_symbolsOut, int *g_countsOut, int n, int* g_totalRuns) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){

		int globalOffset = g_blocksOffset[blockIdx.x];
		int localOffset;

		if (g_backwardMask[i] == 1){

			localOffset = g_scannedBackwardMask[i];
			
			int symbol = g_in[i];

			g_symbolsOut[localOffset + globalOffset] = symbol;
			g_countsOut[localOffset + globalOffset] += -tid;
		}
		if (g_forwardMask[i] == 1){
			localOffset = g_scannedForwardMask[i];
			g_countsOut[localOffset + globalOffset] += tid+1;
		}

		if ((i + 1) == n) {
			*g_totalRuns = localOffset + globalOffset + 1;
		}



	}
}

__global__ void maskKernel(int *g_in, int* g_backwardMask, int* g_forwardMask, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){
		if (tid == 0) 
			g_backwardMask[i] = 1;
		else {
			g_backwardMask[i] = (g_in[i] != g_in[i - 1]);
		}

		if (tid== (blockDim.x-1 ) )
			g_forwardMask[i] = 1;
		else {
			g_forwardMask[i] = (g_in[i] != g_in[i + 1]);
		}

	}
}

void PrintArray(int* arr, int n, int j){
	for (int i = 0; i < n; ++i){
		printf("%d, ", arr[i]);
		if ( (i+1) % j == (0) && i!=0 ){
			printf("| ");
		}
	}
	printf("\n");
}

void verifyCompression(
	int* original, int n, 
	int* compressedSymbols, int* compressedCounts, int totalRuns){

	int* decompressed = new int[n];

	printf("Original Size  : %d\n", n);
	printf("Compressed Size: %d\n", totalRuns*2);


	// decompress.
	int j = 0;
	for (int i = 0; i < totalRuns; ++i){
		int symbol = compressedSymbols[i];
		int count  = compressedCounts[i];

		for (int k = 0; k < count; ++k){
			decompressed[j++] = symbol;
		}
	}

	// verify the compression.
	for (int i = 0; i < n; ++i) {
		if (original[i] != decompressed[i]){
			printf("Decompressed and original not equal at %d, %d != %d\n", i, original[i], decompressed[i]);
		}
	}

	//printf("Decompressed:     ");
//	PrintArray(decompressed, n);




}

int main()
{
	sdkCreateTimer(&timer);

	const int n = 1<<11; // 61, 31

	int* in = new int[n];
	
	// 4, 4, 4, 2, 3, 3, 3, 1, 1, 1, 1, 1, 1, 2, 3, 1 

	/*
	int i = 0;
	in[i++] = 4; in[i++] = 4; in[i++] = 4;// 3
	in[i++] = 2; // 1
	in[i++] = 3; in[i++] = 3; in[i++] = 3; // 3
	in[i++] = 1; in[i++] = 1; in[i++] = 1; 	in[i++] = 9; in[i++] = 1; in[i++] = 1; // 6
	in[i++] = 2; // 1
	in[i++] = 3; // 1
	in[i++] = 1; // 1

	in[i++] = 5; // 1
	in[i++] = 5; // 1
	in[i++] = 5; // 1

	in[i++] = 5; // 1
	in[i++] = 5; // 1
	in[i++] = 3; // 1

	in[i++] = 3; // 1
	in[i++] = 3; // 1
	*/

	srand(1000);

	in[0] = 0;

	for (int i = 1; i < n; ++i) {
		in[i] = in[i - 1] + (rand() % 6 == 0);
	}

	
	CUDA_CHECK(hipSetDevice(0));


	int* symbolsOut = new int[2 * n];
	int* countsOut = new int[2 * n];
	
	int totalRuns = parle(in, n, symbolsOut, countsOut, /*7*/ 1<<8); // 30, 7

	// input: 
	/*printf("Input:            ");
	PrintArray(in, n, 100000);
	*/
	
	verifyCompression(
		in, n,
		symbolsOut, countsOut, totalRuns);
	
	CUDA_CHECK(hipDeviceReset());
	

	printf("DONE\n");
	return 0;
}

int parle(int *in, int n,
	int* h_symbolsOut,
	int* h_countsOut,
	int blockSize){

	int* d_backwardMask;
	int* d_scannedBackwardMask;
	int* d_scannedForwardMask;
	int* d_forwardMask;
	int* d_in;



	// keeps track of the number of runs per block. So d_blocksRunCount[0] is the number of runs for block number 0. 
	int* d_blocksRunCount;
	int* d_blocksOffset;

	int* d_symbolsOut;

	int* d_countsOut;
	int* d_totalRuns; // keeps track of the total number of runs that the data was compressed down to.


	const int BLOCK_SIZE = blockSize;
	const int BLOCK_COUNT = (int)ceil( n / (double)BLOCK_SIZE );
	const int N = BLOCK_COUNT * BLOCK_SIZE;

	/*
	printf("N: %d\n", N);
	printf("n: %d\n", n);
	printf("blocksize: %d\n", BLOCK_SIZE);
	printf("BLOCK_COUNT: %d\n", BLOCK_COUNT);
	*/
	
	int padding = 0; // default padding char is 0.
	if (in[n - 1] == 0){
		padding = 1; // but else use 1. 
	}
	// we use padding if there is not enough input data to fill all the thread blocks. 
	bool usePadding = N != n; // 
	int* h_in = new int[N];

	for (int i = 0; i < N; ++i) {
		if (i < n){
			h_in[i] = in[i];
		}
		else{
			h_in[i] = padding; 
		}
	}
	/*
	printf("use pad: %d", usePadding);

	printf("orig:      ");
	PrintArray(in, n, BLOCK_SIZE);


	printf("padded:      ");	
	PrintArray(h_in, N, BLOCK_SIZE);
	*/


	// allocate resources on device. 
	CUDA_CHECK(hipMalloc((void**)&d_in, N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_backwardMask, N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_forwardMask, N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_scannedBackwardMask, N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_scannedForwardMask, N * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_blocksRunCount, BLOCK_COUNT * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_blocksOffset, BLOCK_COUNT * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_countsOut, 2 * N * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_symbolsOut, 2 * N * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_totalRuns, sizeof(int)));


	// transer input data to device.
	CUDA_CHECK(hipMemcpy(d_in, h_in, N*sizeof(int), hipMemcpyHostToDevice));

	
	// get forward and backward mask. 
	maskKernel<<<BLOCK_COUNT, BLOCK_SIZE>>>(d_in, d_backwardMask, d_forwardMask, N);
	CUDA_CHECK(hipGetLastError());
	
	
	scanKernel << <BLOCK_COUNT, BLOCK_SIZE, 2 * N * sizeof(int) >> >(d_backwardMask, d_scannedBackwardMask, N);
	CUDA_CHECK(hipGetLastError());
	
	scanKernel << <BLOCK_COUNT, BLOCK_SIZE, 2 * N * sizeof(int) >> >(d_forwardMask, d_scannedForwardMask, N);
	CUDA_CHECK(hipGetLastError());

	
	getBlocksRunCountKernel << <BLOCK_COUNT, BLOCK_SIZE >> >(d_scannedBackwardMask, d_blocksRunCount, N);
	CUDA_CHECK(hipGetLastError());
	/*
	printf("block count:%d \n", BLOCK_COUNT);
	printf("allocate this muc:%d \n", 2 * BLOCK_COUNT * sizeof(int));
	*/
	// TODO: there may not be enough thread if there are many blocks!
	//scanKernel2 << <1, BLOCK_COUNT, 2 * BLOCK_COUNT * sizeof(int) >> >(d_blocksRunCount, d_blocksOffset, BLOCK_COUNT);
	scanKernel << <1, BLOCK_COUNT, 2 * BLOCK_COUNT * sizeof(int) >> >(d_blocksRunCount, d_blocksOffset, BLOCK_COUNT);

	CUDA_CHECK(hipGetLastError());
	
	
	
	scatterKernel << <BLOCK_COUNT, BLOCK_SIZE >> >(
		d_backwardMask, d_scannedBackwardMask, 
		d_forwardMask, d_scannedForwardMask,

		d_blocksOffset, d_in, 
		d_symbolsOut, d_countsOut, N, d_totalRuns);
	CUDA_CHECK(hipGetLastError());
	
	
	
	int* h_backwardMask        = new int[N];
	int* h_forwardMask         = new int[N];
	int* h_scannedBackwardMask = new int[N];
	int* h_scannedForwardMask  = new int[N];

	int* h_blocksRunCount      = new int[BLOCK_COUNT];
	int* h_blocksOffset        = new int[BLOCK_COUNT];

	int h_totalRuns;

	//CUDA_CHECK(hipDeviceSynchronize());

	
	CUDA_CHECK(hipMemcpy(h_backwardMask, d_backwardMask, N*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_forwardMask, d_forwardMask, N*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_scannedBackwardMask, d_scannedBackwardMask, N*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_scannedForwardMask, d_scannedForwardMask, N*sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipMemcpy(h_blocksRunCount, d_blocksRunCount, BLOCK_COUNT*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_blocksOffset, d_blocksOffset, BLOCK_COUNT*sizeof(int), hipMemcpyDeviceToHost));
	
	
	CUDA_CHECK(hipMemcpy(h_symbolsOut, d_symbolsOut, 2*n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_countsOut, d_countsOut, 2 *n*sizeof(int), hipMemcpyDeviceToHost));
	
	CUDA_CHECK(hipMemcpy(&h_totalRuns, d_totalRuns, sizeof(int), hipMemcpyDeviceToHost));
	
	if (usePadding) {
		// if we use padding, then the last run will just be the compressed padding characters.
		// so skip that last run:
		--h_totalRuns;
	
	}
	
	
	/*
	printf("Backward:         ");
	PrintArray(h_backwardMask, N, BLOCK_SIZE);
	
	printf("Forward:          ");
	PrintArray(h_forwardMask, N, BLOCK_SIZE);

	printf("Scanned Backward: ");
	PrintArray(h_scannedBackwardMask, N, BLOCK_SIZE);

	printf("Scanned Forward:  ");
	PrintArray(h_scannedForwardMask, N, BLOCK_SIZE);
	
	printf("h_blocksRunCount: ");
	PrintArray(h_blocksRunCount, BLOCK_COUNT, BLOCK_SIZE);

	printf("h_blocksOffset:   ");
	PrintArray(h_blocksOffset, BLOCK_COUNT, BLOCK_SIZE);
	
	printf("h_symbolsOut:     ");
	PrintArray(h_symbolsOut, h_totalRuns);

	printf("h_countsOut:      ");
	PrintArray(h_countsOut, h_totalRuns);



	printf("h_totalRuns:      %d\n", h_totalRuns);
	*/
	
	return h_totalRuns;
	// TODO: hipFree.
	
	return 0;
}

/*
// Helper function for using CUDA to add vectors in parallel.
void scan(int *in, int* out, int n)
{
	int *inBuffer = 0;
	int *outBuffer = 0;

	const unsigned int BLOCK_SIZE = 1024;
 

	CUDA_CHECK(hipMalloc((void**)&inBuffer, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&outBuffer, n * sizeof(int)));

	CUDA_CHECK(hipMemcpy(inBuffer, in, n * sizeof(int), hipMemcpyHostToDevice));
	
	// warm up.
	scanKernel<<<1, n, sizeof(int)*n*2 >> >( inBuffer, outBuffer, n);
	
	
	for (int i = 0; i < 20; ++i){
		hipDeviceSynchronize();
		sdkStartTimer(&timer);
	
		scanKernel << <1, n, sizeof(int)*n * 2 >> >(inBuffer, outBuffer, n);

		// Copy output vector from GPU buffer to host memory.
		hipDeviceSynchronize();
		sdkStopTimer(&timer);
	}
		
	double reduceTime = sdkGetAverageTimerValue(&timer) * 1e-3;	
	printf("average: %.4f GB/s, Time = %.5f s\n", 1.0e-9 * ( (double)  (sizeof( int)*n)/reduceTime  ), reduceTime     );
	

	
	CUDA_CHECK(hipMemcpy(out, outBuffer, n * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	hipFree(inBuffer);
	hipFree(outBuffer);
}
*/