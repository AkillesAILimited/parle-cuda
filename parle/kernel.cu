#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "helper_timer.h"
#include "hip/hip_runtime_api.h"

void scan(int *in, int* out, int n);

void rle(int *h_in, int n,
	int* h_symbolsOut,
	int* h_countsOut);


__global__ void scanKernel(int *g_idata, int *g_odata, int n) {
	extern __shared__ float temp[]; // size is 2*n
	int tid = threadIdx.x;
	int i = blockIdx.x * blockDim.x + tid;
	int pout = 0;
	int pin = 1;

	temp[pout*n + i] = (tid > 0) ? g_idata[i-1] : 0;
	//temp[pout*n + i] = g_idata[i];

	__syncthreads();
	
	for (int offset = 1; offset < blockDim.x; offset *= 2)
	{
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (tid >= offset)
			temp[pout*n + i] = temp[pin*n + i] +  temp[pin*n + i - offset];
		else
			temp[pout*n + i] = temp[pin*n + i];
		__syncthreads();
	}
	
	g_odata[i] = temp[pout*n + i]; // write output

}

__global__ void getBlocksRunCountKernel(int *g_idata, int *g_odata, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){

		if (tid == (blockDim.x - 1)){
			g_odata[blockIdx.x] = g_idata[i]+1;
		}

	}
}

__global__ void scatterKernel(
	int *g_backwardMask, int* g_scannedBackwardMask, 
	int *g_forwardMask, int* g_scannedForwardMask,
	int *g_blocksOffset, int *g_in, 
	int *g_symbolsOut, int *g_countsOut, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){

		int globalOffset = g_blocksOffset[blockIdx.x];

		if (g_backwardMask[i] == 1){

			int localOffset = g_scannedBackwardMask[i];
			
			int symbol = g_in[i];

			g_symbolsOut[localOffset + globalOffset] = symbol;
			g_countsOut[localOffset + globalOffset] += -tid;
		}
		if (g_forwardMask[i] == 1){
			int localOffset = g_scannedForwardMask[i];
			g_countsOut[localOffset + globalOffset] += tid+1;
		}
	}
}

__global__ void maskKernel(int *g_in, int* g_backwardMask, int* g_forwardMask, int n) {

	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;

	if (i < n){
		if (tid == 0) 
			g_backwardMask[i] = 1;
		else {
			g_backwardMask[i] = (g_in[i] != g_in[i - 1]);
		}

		if (tid== (blockDim.x-1 ) )
			g_forwardMask[i] = 1;
		else {
			g_forwardMask[i] = (g_in[i] != g_in[i + 1]);
		}

	}
}

void PrintArray(int* arr, int n){
	for (int i = 0; i < n; ++i){
		printf("%d, ", arr[i]);
		if (i == (n/2-1) ){
			printf("| ");
		}
	}
	printf("\n");
}


int main()
{
	sdkCreateTimer(&timer);

	const int n = 16;

	int* in = new int[n];
	
	// 4, 4, 4, 2, 3, 3, 3, 1, 1, 1, 1, 1, 1, 2, 3, 1 

	int i = 0;
	in[i++] = 4; in[i++] = 4; in[i++] = 4;// 3
	in[i++] = 2; // 1
	in[i++] = 3; in[i++] = 3; in[i++] = 3; // 3
	in[i++] = 1; in[i++] = 1; in[i++] = 1; 	in[i++] = 9; in[i++] = 1; in[i++] = 1; // 6
	in[i++] = 2; // 1
	in[i++] = 3; // 1
	in[i++] = 1; // 1

	
	CUDA_CHECK(hipSetDevice(0));


	int* symbolsOut = new int[2 * n];
	int* countsOut = new int[2 * n];

	rle(in, n, symbolsOut, countsOut);

	// input: 
	printf("Input:            ");
	PrintArray(in, n);
	
	CUDA_CHECK(hipDeviceReset());


	printf("DONE\n");
	return 0;
}

void rle(int *h_in, int n,
	int* h_symbolsOut,
	int* h_countsOut){

	int* d_backwardMask;
	int* d_scannedBackwardMask;
	int* d_scannedForwardMask;
	int* d_forwardMask;
	int* d_in;


	// keeps track of the number of runs per block. So d_blocksRunCount[0] is the number of runs for block number 0. 
	int* d_blocksRunCount;
	int* d_blocksOffset;

	int* d_symbolsOut;

	int* d_countsOut;



	const int BLOCK_COUNT = 2;
	const int BLOCK_SIZE = n / BLOCK_COUNT;

	// allocate resources on device. 
	CUDA_CHECK(hipMalloc((void**)&d_in, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_backwardMask, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_forwardMask, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_scannedBackwardMask, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_scannedForwardMask, n * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_blocksRunCount, BLOCK_COUNT * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_blocksOffset, BLOCK_COUNT * sizeof(int)));

	CUDA_CHECK(hipMalloc((void**)&d_countsOut, 2 * n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&d_symbolsOut, 2 * n * sizeof(int)));


	// transer input data to device.
	CUDA_CHECK(hipMemcpy(d_in, h_in, n*sizeof(int), hipMemcpyHostToDevice));

	// get forward and backward mask. 
	maskKernel<<<BLOCK_COUNT, BLOCK_SIZE>>>(d_in, d_backwardMask, d_forwardMask, n);

	scanKernel << <BLOCK_COUNT, BLOCK_SIZE, 2 * n * sizeof(int) >> >(d_backwardMask, d_scannedBackwardMask, n);
	scanKernel << <BLOCK_COUNT, BLOCK_SIZE, 2 * n * sizeof(int) >> >(d_forwardMask, d_scannedForwardMask, n);


	getBlocksRunCountKernel << <BLOCK_COUNT, BLOCK_SIZE >> >(d_scannedBackwardMask, d_blocksRunCount, n);

	scanKernel << <1, BLOCK_COUNT, 2 * BLOCK_COUNT * sizeof(int) >> >(d_blocksRunCount, d_blocksOffset, n);



	scatterKernel << <BLOCK_COUNT, BLOCK_SIZE >> >(
		d_backwardMask, d_scannedBackwardMask, 
		d_forwardMask, d_scannedForwardMask,

		d_blocksOffset, d_in, 
		d_symbolsOut, d_countsOut, n);


	
	int* h_backwardMask        = new int[n];
	int* h_forwardMask         = new int[n];
	int* h_scannedBackwardMask = new int[n];
	int* h_scannedForwardMask  = new int[n];

	int* h_blocksRunCount      = new int[BLOCK_COUNT];
	int* h_blocksOffset        = new int[BLOCK_COUNT];

	CUDA_CHECK(hipMemcpy(h_backwardMask, d_backwardMask, n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_forwardMask, d_forwardMask, n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_scannedBackwardMask, d_scannedBackwardMask, n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_scannedForwardMask, d_scannedForwardMask, n*sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipMemcpy(h_blocksRunCount, d_blocksRunCount, BLOCK_COUNT*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_blocksOffset, d_blocksOffset, BLOCK_COUNT*sizeof(int), hipMemcpyDeviceToHost));

	
	CUDA_CHECK(hipMemcpy(h_symbolsOut, d_symbolsOut, 2*n*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_countsOut, d_countsOut, 2 * n*sizeof(int), hipMemcpyDeviceToHost));
	

	printf("Backward:         ");
	PrintArray(h_backwardMask, n);

	printf("Forward:          ");
	PrintArray(h_forwardMask, n);

	printf("Scanned Backward: ");
	PrintArray(h_scannedBackwardMask, n);

	printf("Scanned Forward:  ");
	PrintArray(h_scannedForwardMask, n);

	printf("h_blocksRunCount: ");
	PrintArray(h_blocksRunCount, BLOCK_COUNT);

	printf("h_blocksOffset:   ");
	PrintArray(h_blocksOffset, BLOCK_COUNT);

	printf("h_symbolsOut:     ");
	PrintArray(h_symbolsOut, 10);

	printf("h_countsOut:      ");
	PrintArray(h_countsOut, 10);

	// TODO: hipFree.
}

/*
// Helper function for using CUDA to add vectors in parallel.
void scan(int *in, int* out, int n)
{
	int *inBuffer = 0;
	int *outBuffer = 0;

	const unsigned int BLOCK_SIZE = 1024;
 

	CUDA_CHECK(hipMalloc((void**)&inBuffer, n * sizeof(int)));
	CUDA_CHECK(hipMalloc((void**)&outBuffer, n * sizeof(int)));

	CUDA_CHECK(hipMemcpy(inBuffer, in, n * sizeof(int), hipMemcpyHostToDevice));
	
	// warm up.
	scanKernel<<<1, n, sizeof(int)*n*2 >> >( inBuffer, outBuffer, n);
	
	
	for (int i = 0; i < 20; ++i){
		hipDeviceSynchronize();
		sdkStartTimer(&timer);
	
		scanKernel << <1, n, sizeof(int)*n * 2 >> >(inBuffer, outBuffer, n);

		// Copy output vector from GPU buffer to host memory.
		hipDeviceSynchronize();
		sdkStopTimer(&timer);
	}
		
	double reduceTime = sdkGetAverageTimerValue(&timer) * 1e-3;	
	printf("average: %.4f GB/s, Time = %.5f s\n", 1.0e-9 * ( (double)  (sizeof( int)*n)/reduceTime  ), reduceTime     );
	

	
	CUDA_CHECK(hipMemcpy(out, outBuffer, n * sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	hipFree(inBuffer);
	hipFree(outBuffer);
}
*/